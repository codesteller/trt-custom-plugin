#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cassert>
#include <cstring>
#include <vector>

#include "NvInfer.h"
#include "geluPlugin.h"
#include "bertCommon.h"
#include "common.h"
#include "serialize.hpp"

using namespace nvinfer1;

namespace GELU
{

// constants for approximating the normal cdf
constexpr float A = 0.5;
constexpr float B = 0.7978845608028654;   // sqrt(2.0/M_PI)
constexpr float C = 0.035677408136300125; // 0.044715 * sqrt(2.0/M_PI)

template <typename T, unsigned TPB>
__global__ void geluKernel(const T a, const T b, const T c, int n, const T* input, T* output)
{

    const int idx = blockIdx.x * TPB + threadIdx.x;

    if (idx < n)
    {
        const T in = input[idx];
        const T cdf = a + a * tanh(in * (c * in * in + b));
        output[idx] = in * cdf;
    }
}

inline int computeGelu(hipStream_t stream, int n, const float* input, float* output)
{

    constexpr int blockSize = 256;
    const int gridSize = (n + blockSize - 1) / blockSize;
    geluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, n, input, output);

    CHECK(hipPeekAtLastError());
    return 0;
}

inline int computeGelu(hipStream_t stream, int n, const half* input, half* output)
{
    const int blockSize = 256;

    if (0 == (n & 1))
    {
        const int n2 = n / 2;

        const int gridSize = (n2 + blockSize - 1) / blockSize;
        const half2 A2 = __floats2half2_rn(A, A);
        const half2 B2 = __floats2half2_rn(B, B);
        const half2 C2 = __floats2half2_rn(C, C);
        const half2* input2 = reinterpret_cast<const half2*>(input);
        half2* output2 = reinterpret_cast<half2*>(output);
        geluKernel<half2, blockSize><<<gridSize, blockSize, 0, stream>>>(A2, B2, C2, n2, input2, output2);
    }
    else
    {
        const int gridSize = (n + blockSize - 1) / blockSize;
        geluKernel<half, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, n, input, output);
    }

    CHECK(hipPeekAtLastError());
    return 0;
}

template <typename T, int TPB>
__global__ void geluBiasKernel(const T a, const T b, const T c, T* output, const T* input, const T* bias, const int ld)
{

    const int offset = blockIdx.x * ld;

    for (int it = threadIdx.x; it < ld; it += TPB)
    {
        const int idx = it + offset;
        const T in = input[idx] + bias[it];
        const T cdf = a + a * tanh(in * (c * in * in + b));
        output[idx] = in * cdf;
    }
}

void computeGeluBias(
    float* output, const float* input, const float* bias, const int ld, const int cols, hipStream_t stream)
{
    geluBiasKernel<float, 256><<<cols, 256, 0, stream>>>(A, B, C, output, input, bias, ld);
    CHECK(hipPeekAtLastError());
}

void computeGeluBias(
    half* output, const half* input, const half* bias, const int ld, const int cols, hipStream_t stream)
{
    if (ld & 1)
    {
        geluBiasKernel<half, 256><<<cols, 256, 0, stream>>>(A, B, C, output, input, bias, ld);
    }
    else
    {

        const half2 A2 = __floats2half2_rn(A, A);
        const half2 B2 = __floats2half2_rn(B, B);
        const half2 C2 = __floats2half2_rn(C, C);
        const int ld2 = ld / 2;
        const half2* input2 = reinterpret_cast<const half2*>(input);
        const half2* bias2 = reinterpret_cast<const half2*>(bias);
        half2* output2 = reinterpret_cast<half2*>(output);
        geluBiasKernel<half2, 256><<<cols, 256, 0, stream>>>(A2, B2, C2, output2, input2, bias2, ld2);
    }

    CHECK(hipPeekAtLastError());
}

/////////////////////////////////

namespace
{
static const char* GELU_PLUGIN_VERSION{"1"};
static const char* GELU_PLUGIN_NAME{"CustomGeluPluginDynamic"};
} // namespace

// Static class fields initialization
PluginFieldCollection GeluPluginDynamicCreator::mFC{};
std::vector<PluginField> GeluPluginDynamicCreator::mPluginAttributes;

REGISTER_TENSORRT_PLUGIN(GeluPluginDynamicCreator);

GeluPluginDynamic::GeluPluginDynamic(const std::string name, const DataType type)
    : mLayerName(name)
    , mType(type)
    , mHasBias(false)
    , mLd(0)
{
    mBias.values = nullptr;
    mBias.count = 0;
}

GeluPluginDynamic::GeluPluginDynamic(const std::string name, const DataType type, const Weights B)
    : mLayerName(name)
    , mType(type)
    , mHasBias(true)
    , mBias(B)
    , mLd(B.count)
{
}

GeluPluginDynamic::GeluPluginDynamic(const std::string name, const void* data, size_t length)
    : mLayerName(name)
{
    gLogVerbose << "Starting to deserialize GELU plugin" << std::endl;
    deserialize_value(&data, &length, &mType);
    deserialize_value(&data, &length, &mLd);
    deserialize_value(&data, &length, &mHasBias);

    gLogVerbose << "Deserialized parameters: mLd: " << mLd << ", mHasBias: " << mHasBias << std::endl;
    if (mHasBias)
    {
        const char* d = static_cast<const char*>(data);
        gLogVerbose << "Deserializing Bias" << std::endl;
        if (mLd <= 0)
        {
            gLogError << "Gelu+bias: deserialization inconsistent. HasBias but mLd is 0" << std::endl;
        }
        const size_t wordSize = samplesCommon::getElementSize(mType);
        mBiasDev = deserToDev<char>(d, mLd * wordSize);
    }
    gLogVerbose << "Finished deserializing GELU plugin" << std::endl;
    mBias.values = nullptr;
    mBias.count = mLd;
}
// IPluginV2DynamicExt Methods
nvinfer1::IPluginV2DynamicExt* GeluPluginDynamic::clone() const
{
    if (mHasBias)
    {
        return new GeluPluginDynamic(mLayerName, mType, mBias);
    }
    return new GeluPluginDynamic(mLayerName, mType);
}

nvinfer1::DimsExprs GeluPluginDynamic::getOutputDimensions(
    int outputIndex, const nvinfer1::DimsExprs* inputs, int nbInputs, nvinfer1::IExprBuilder& exprBuilder)
{
    return inputs[0];
}

bool GeluPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* inOut, int nbInputs, int nbOutputs)
{

    const PluginTensorDesc& input = inOut[0];
    if (pos == 0)
    {
        return (input.type == mType) && (input.format == TensorFormat::kLINEAR);
    }
    if (pos == 1)
    {
        const PluginTensorDesc& output = inOut[1];
        return (input.type == output.type) && (output.format == TensorFormat::kLINEAR);
    }
    return false;
}

void GeluPluginDynamic::configurePlugin(const nvinfer1::DynamicPluginTensorDesc* in, int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* out, int nbOutputs)
{
    assert(mType == in[0].desc.type);
}

size_t GeluPluginDynamic::getWorkspaceSize(const nvinfer1::PluginTensorDesc* inputs, int nbInputs,
    const nvinfer1::PluginTensorDesc* outputs, int nbOutputs) const
{
    return 0;
}
int GeluPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* inputDesc,
    const nvinfer1::PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace,
    hipStream_t stream)
{
    const int inputVolume = volume(inputDesc[0].dims);

    int status = -1;

    // Our plugin outputs only one tensor
    // Launch CUDA kernel wrapper and save its return value
    if (mType == DataType::kFLOAT)
    {
        const float* input = static_cast<const float*>(inputs[0]);
        float* output = static_cast<float*>(outputs[0]);
        if (mHasBias)
        {
            const float* bias = reinterpret_cast<float*>(mBiasDev);
            const int cols = inputVolume / mLd;
            const int rows = mLd;
            computeGeluBias(output, input, bias, rows, cols, stream);
        }
        else
        {
            status = computeGelu(stream, inputVolume, input, output);
        }
    }
    else if (mType == DataType::kHALF)
    {
        const half* input = static_cast<const half*>(inputs[0]);

        half* output = static_cast<half*>(outputs[0]);

        if (mHasBias)
        {
            const half* bias = reinterpret_cast<half*>(mBiasDev);
            const int cols = inputVolume / mLd;
            const int rows = mLd;
            computeGeluBias(output, input, bias, rows, cols, stream);
        }
        else
        {
            status = computeGelu(stream, inputVolume, input, output);
        }
    }
    else
    {
        assert(false);
    }

    return status;
}

// IPluginV2Ext Methods
nvinfer1::DataType GeluPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
{
    assert(index == 0);
    assert(inputTypes[0] == DataType::kFLOAT || inputTypes[0] == DataType::kHALF);
    return inputTypes[0];
}

// IPluginV2 Methods

const char* GeluPluginDynamic::getPluginType() const
{
    return GELU_PLUGIN_NAME;
}

const char* GeluPluginDynamic::getPluginVersion() const
{
    return GELU_PLUGIN_VERSION;
}

int GeluPluginDynamic::getNbOutputs() const
{
    return 1;
}

int GeluPluginDynamic::initialize()
{
    gLogVerbose << "GELU init start" << std::endl;
    if (mHasBias && mBias.values)
    {
        // target size
        const size_t wordSize = samplesCommon::getElementSize(mType);
        const size_t nbBytes = mBias.count * wordSize;
        CHECK(hipMalloc(&mBiasDev, nbBytes));

        if (mType == DataType::kFLOAT)
        {
            convertAndCopyToDevice(mBias, reinterpret_cast<float*>(mBiasDev));
        }
        else
        {
            convertAndCopyToDevice(mBias, reinterpret_cast<half*>(mBiasDev));
        }
    }
    gLogVerbose << "GELU init done" << std::endl;
    return 0;
}

void GeluPluginDynamic::terminate()
{
    if (mHasBias)
    {
        CHECK(hipFree(mBiasDev));
    }
}

size_t GeluPluginDynamic::getSerializationSize() const
{
    const size_t wordSize = samplesCommon::getElementSize(mType);
    const size_t biasSize = mHasBias ? mLd * wordSize : 0;
    return sizeof(mType) + sizeof(mHasBias) + sizeof(mLd) + biasSize;
}

void GeluPluginDynamic::serialize(void* buffer) const
{
    serialize_value(&buffer, mType);
    serialize_value(&buffer, mLd);
    serialize_value(&buffer, mHasBias);
    if (mHasBias)
    {
        char *d = static_cast<char*>(buffer);
        const size_t wordSize = samplesCommon::getElementSize(mType);
        const size_t biasSize = mHasBias ? mLd * wordSize : 0;
        if (biasSize <= 0)
        {
            gLogError << "Gelu+bias: bias size inconsistent" << std::endl;
        }
        serFromDev(d, mBiasDev, mLd * wordSize);
    }
}

void GeluPluginDynamic::destroy()
{
    // This gets called when the network containing plugin is destroyed
    delete this;
}

void GeluPluginDynamic::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* GeluPluginDynamic::getPluginNamespace() const
{
    return mNamespace.c_str();
}

///////////////

GeluPluginDynamicCreator::GeluPluginDynamicCreator()
{

    // Fill PluginFieldCollection with PluginField arguments metadata
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* GeluPluginDynamicCreator::getPluginName() const
{
    return GELU_PLUGIN_NAME;
}

const char* GeluPluginDynamicCreator::getPluginVersion() const
{
    return GELU_PLUGIN_VERSION;
}

const PluginFieldCollection* GeluPluginDynamicCreator::getFieldNames()
{
    return &mFC;
}

IPluginV2* GeluPluginDynamicCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
{

    Weights bias{DataType::kFLOAT, nullptr, 0};
    int typeId = -1;
    for (int i = 0; i < fc->nbFields; i++)
    {
        std::string field_name(fc->fields[i].name);

        if (field_name.compare("type_id") == 0)
        {
            typeId = *static_cast<const int*>(fc->fields[i].data);
            gLogVerbose << "Building typeId: " << typeId << std::endl;
        }

        if (field_name.compare("bias") == 0)
        {
            gLogVerbose << "Building bias...\n";
            bias.values = fc->fields[i].data;
            bias.count = fc->fields[i].length;
            bias.type = fieldTypeToDataType(fc->fields[i].type);
        }
    }

    if (typeId < 0 || typeId > 3)
    {
        gLogError << "GELU: invalid typeId " << typeId << std::endl;
        return nullptr;
    }
    DataType type = static_cast<DataType>(typeId);
    gLogVerbose << "Creating GeluPluginDynamic...\n";
    if (bias.values == nullptr)
    {
        return new GeluPluginDynamic(name, type);
    }

    return new GeluPluginDynamic(name, type, bias);
}

IPluginV2* GeluPluginDynamicCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
{
    // This object will be deleted when the network is destroyed, which will
    // call GeluPluginDynamic::destroy()
    return new GeluPluginDynamic(name, serialData, serialLength);
}

void GeluPluginDynamicCreator::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* GeluPluginDynamicCreator::getPluginNamespace() const
{
    return mNamespace.c_str();
}
}
